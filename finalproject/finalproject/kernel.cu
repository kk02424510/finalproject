#include <stdio.h>
#include <stdlib.h>
#include <cv.h>

#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>

int main()
{

	IplImage* img1 = cvLoadImage("1.jpg");
	IplImage* img2 = cvLoadImage("2.jpg");

	int i, j, k,a = 0,b = 0, index ,index2;
	int height, width, step, channels;
	int height2, width2, step2, channels2;
	int flag = false ;

	uchar* data1, * data2;


	// get the image data
	height = img1->height;
	width = img1->width;
	step = img1->widthStep;
	height2 = img2->height;
	width2 = img2->width;
	step2 = img2->widthStep;
	channels = img1->nChannels;
	channels2 = img2->nChannels;
	data1 = (uchar*)img1->imageData;
	data2 = (uchar*)img2->imageData;

	printf("Processing a %dx%d image with %d channels, and %d steps.\n", height, width, channels, step);


	// create a window
	cvNamedWindow("mainWin", CV_WINDOW_AUTOSIZE);
	cvMoveWindow("mainWin", 100, 100);

	// mix the image
	for (i = 0; i < height; i++)
	{
		for (j = 0; j < width; j++)
		{
			for (k = 0; k < channels; k++)
			{
				index = i * step + j * channels + k;

				if (k % 3 == 0)
				{
					if (data1[index] < 30 && data1[index + 1] < 30 && data1[index + 2] < 30)
					{	
						
						index2 = a * step2 + b * channels2;
						data1[index] = data2[index2];
						data1[index+1] = data2[index2+1];
						data1[index+2] = data2[index2+2];
						b ++ ;
						flag = true;
					}
				}
			}			
			
		}
		if (flag)
		{
			b = 0, a++;
			flag = false;
		}
		

	}

	// show the image
	cvShowImage("mainWin", img1);

	// wait for a key
	cvWaitKey(0);

	// release the image
	cvReleaseImage(&img1);

	return 0;
}
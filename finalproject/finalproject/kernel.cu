#include <stdio.h>
#include <stdlib.h>
#include <cv.h>

#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>

uchar blue[700][825];
uchar green[700][825];
uchar red[700][825];

int main()
{

	IplImage* img1 = cvLoadImage("1.jpg");
	IplImage* img2 = cvLoadImage("2.jpg");

	int i, j, index;
	int flag = false;
	int x = 0, y = 0;
	int height, width, step, channels;
	uchar* data1, * data2;
	

	// get the image data
	height = img1->height;
	width = img1->width;
	step = img1->widthStep;
	channels = img1->nChannels;
	data1 = (uchar*)img1->imageData;
	data2 = (uchar*)img2->imageData;

	printf("Processing a %dx%d image with %d channels, and %d steps.\n", height, width, channels, step);


	// create a window
	cvNamedWindow("img1", CV_WINDOW_AUTOSIZE);
	cvNamedWindow("img2", CV_WINDOW_AUTOSIZE);
	cvMoveWindow("mainWin", 100, 100);

	// load rgb
	for (i = 0; i < height; i++)
	{
		for (j = 0; j < step; j=j+3)
		{
			blue[i][(int)(j / 3)] = img1->imageData[i * img1->widthStep + j];
			green[i][(int)(j / 3)] = img1->imageData[i * img1->widthStep + j + 1];
			red[i][(int)(j / 3)] = img1->imageData[i * img1->widthStep + j + 2];					
		}
	}

	for (i = 0; i < height; i++)
	{
		for (j = 0; j < step; j = j + 3)
		{
			if (blue[i][(int)(j / 3)] < 30 && green[i][(int)(j / 3)] <30 && red[i][(int)(j / 3)] <30)
			{		
				if (x < img2->height && y < img2->widthStep)
				{
					data1[i * img1->widthStep + j] = data2[x * img2->widthStep + y];
					data1[i * img1->widthStep + j + 1] = data2[x * img2->widthStep + y + 1];
					data1[i * img1->widthStep + j + 2] = data2[x * img2->widthStep + y + 2];
					y = y + 3;
					flag = true;
				}
			}
		}
		if (flag) {
			y = 0;
			x = x + 1;
			flag = false;
		}
	}

	// show the image
	cvShowImage("img1", img1);
	cvShowImage("img2", img2);


	// wait for a key
	cvWaitKey(0);

	// release the image
	cvReleaseImage(&img1);

	return 0;
}